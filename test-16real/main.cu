#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include "wmma-common.h"
#include "matrix-utility.h"

#define TCU_MAT 1920
#define RUN     10
#define BLKSIZE 1024
#define MAT_PER_BLOCK 160

int main(int argc, char **argv)
{
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float elapsed;

  half *h_a_tcu;
  half *h_b_tcu;
  half *h_c_tcu;
  cpuAllocTCUMatrixHalf(&h_a_tcu, TCU_MAT);
  cpuAllocTCUMatrixHalf(&h_b_tcu, TCU_MAT);
  cpuAllocTCUMatrixHalf(&h_c_tcu, TCU_MAT);
  fillZeroTCUMatrixHalf(h_a_tcu, TCU_MAT);
  fillZeroTCUMatrixHalf(h_b_tcu, TCU_MAT);
  fillZeroTCUMatrixHalf(h_c_tcu, TCU_MAT);
  fillTCUMatrixHalf(h_a_tcu, TCU_MAT);
  fillTCUMatrixHalf(h_b_tcu, TCU_MAT);

  half *d_a_tcu;
  half *d_b_tcu;
  half *d_c_tcu;
  gpuAllocTCUMatrixHalf((void **)&d_a_tcu, TCU_MAT);
  gpuAllocTCUMatrixHalf((void **)&d_b_tcu, TCU_MAT);
  gpuAllocTCUMatrixHalf((void **)&d_c_tcu, TCU_MAT);
  copyHDTCUMatrixHalf(d_a_tcu, h_a_tcu, TCU_MAT);
  copyHDTCUMatrixHalf(d_b_tcu, h_b_tcu, TCU_MAT);
  copyHDTCUMatrixHalf(d_c_tcu, h_c_tcu, TCU_MAT);

  hipEventRecord(start, 0);

  dim3 grid_tcu(TCU_MAT/MAT_PER_BLOCK);
  dim3 block_tcu(BLKSIZE);

  for (unsigned i = 0; i < RUN; i++) {
    dot_wmma16x16<<<grid_tcu, block_tcu>>>(d_a_tcu, d_b_tcu, d_c_tcu, TCU_MAT);
    hipDeviceSynchronize();
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed, start, stop);
  elapsed /= 1000.0f;
  printf("TCU Version: %fs\n", elapsed/RUN);

  copyDHTCUMatrixHalf(h_c_tcu, d_c_tcu, TCU_MAT);

  for (unsigned i = 0; i < 16; i++) {
    for (unsigned j = 0; j < 16; j++) {
      printf("%.1f\t", __half2float(h_a_tcu[j+i*16]));
    }
    printf("\n");
  }

  printf("\n Second\n");

  for (unsigned i = 0; i < 16; i++) {
    for (unsigned j = 0; j < 16; j++) {
      printf("%.1f\t", __half2float((h_a_tcu+256*383)[j+i*16]));
    }
    printf("\n");
  }
  hipFree(d_a_tcu);
  hipFree(d_b_tcu);
  hipFree(d_c_tcu);
  free(h_a_tcu);
  free(h_b_tcu);
  free(h_c_tcu);


  ////////// End TCU version////////////////////////////////////////////
  //////////////////////////////////////////////////////////////////////

  half *h_a;
  half *h_b;
  half *h_c;
  cpuAllocMatrixHalf(&h_a, 3, 3, TCU_MAT);
  cpuAllocMatrixHalf(&h_b, 3, 1, TCU_MAT);
  cpuAllocMatrixHalf(&h_c, 3, 1, TCU_MAT);
  fillMatrixHalf(h_a, 3, 3, TCU_MAT);
  fillMatrixHalf(h_b, 3, 1, TCU_MAT);
  fillMatrixHalf(h_c, 3, 1, TCU_MAT);

  //printf("Before\n");
  //for (unsigned i = 0; i < 12; i++) {
  //  if (i % 3 == 0) printf("\n");
  //  printf("%f ", __half2float(h_c[i]));
  //}

  half *d_a;
  half *d_b;
  half *d_c;

  gpuAllocMatrixHalf((void **)&d_a, 3, 3, TCU_MAT);
  gpuAllocMatrixHalf((void **)&d_b, 3, 1, TCU_MAT);
  gpuAllocMatrixHalf((void **)&d_c, 3, 1, TCU_MAT);
  copyHDMatrixHalf(d_a, h_a, 3, 3, TCU_MAT);
  copyHDMatrixHalf(d_b, h_b, 3, 1, TCU_MAT);
  copyHDMatrixHalf(d_c, h_c, 3, 1, TCU_MAT);

  hipEventRecord(start, 0);

  dim3 grid((TCU_MAT+BLKSIZE-1)/BLKSIZE);
  dim3 block(BLKSIZE);

  for (unsigned i = 0; i < RUN; i++) {
    mat_vec_mul<<<grid, block>>>(d_a, d_b, d_c, TCU_MAT);
    hipDeviceSynchronize();
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed, start, stop);
  elapsed /= 1000.0f;
  printf("Normal Version: %fs\n", elapsed/RUN);

  copyDHMatrixHalf(h_c, d_c, 3, 1, TCU_MAT);

  //printf("After:\n");
  //for (unsigned i = 0; i < 12; i++) {
  //  if (i % 3 == 0) printf("\n");
  //  printf("%f ", __half2float(h_c[i]));
  //}

  free(h_a);
  free(h_b);
  free(h_c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  //////////////////////// Test cublas full matrix /////////////////////////
  half *d_a_c;
  half *d_b_c;
  half *d_c_c;
  hipMalloc((void **)&d_a_c, sizeof(half)*5760*5760);
  hipMalloc((void **)&d_b_c, sizeof(half)*5760*5760);
  hipMalloc((void **)&d_c_c, sizeof(half)*5760*5760);

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
  half alpha = __float2half(1.0f);
  half beta = __float2half(0.0f);

  hipEventRecord(start, 0);

  hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 5760, 5760, 5760, &alpha, d_a_c, HIP_R_16F, 5760, d_b_c, HIP_R_16F, 5760, &beta, d_c_c, HIP_R_16F, 5760, HIP_R_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed, start, stop);
  elapsed /= 1000.0f;
  printf("Cublas Version: %fs", elapsed);

  hipFree(d_a_c);
  hipFree(d_b_c);
  hipFree(d_c_c);
  return EXIT_SUCCESS;
}
