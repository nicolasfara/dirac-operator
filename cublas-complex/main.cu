#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_complex.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define BLKSIZE 1024
#define ALIGN 128

__device__ bool isValid;

using namespace std;

void inline checkError(hipblasStatus_t status, const char *msg)
{
  if (status != HIPBLAS_STATUS_SUCCESS)
  {
    printf("%s", msg);
    exit(EXIT_FAILURE);
  }
}

__global__ void fillMatrix(__restrict hipFloatComplex * const mat, const unsigned size)
{
  const unsigned i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i < size) {
    mat[i] = make_hipFloatComplex((float) i, 3.0f);
  }
}

__global__ void checkMatrix(const __restrict hipFloatComplex * const m1, const __restrict hipFloatComplex * const m2, const unsigned size, bool *isValid)
{

  const unsigned i = threadIdx.x + blockIdx.x * blockDim.x;
  *isValid = true;

  if (i < size) {
    if (fabsf(hipCrealf(m1[i]) - hipCrealf(m2[i]) > 1e-3)) {
      *isValid = false;
    }

    if (fabsf(hipCimagf(m1[i]) - hipCimagf(m2[i]) > 1e-3)) {
      *isValid = false;
    }
  }

}

int main(int argc, char **argv)
{
  // cuBLAS initializzation
  hipblasHandle_t handle;
  hipblasStatus_t stat;
  checkError(hipblasCreate(&handle), "hipblasCreate() error!\n");
  checkError(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH), "hipblasSetMathMode() error!\n");

  // Cuda event setup
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float elapsed;

  size_t M = 2048;
  size_t K = 4096;
  size_t N = 2048;
  size_t A_size = M * K;
  size_t B_size = K * N;
  size_t C_size = M * N;

  hipFloatComplex *A_h;
  hipFloatComplex *B_h;
  hipFloatComplex *C_h;

  // Host allocation matrix
  posix_memalign((void **)&A_h, ALIGN, sizeof(hipFloatComplex) * A_size);
  posix_memalign((void **)&B_h, ALIGN, sizeof(hipFloatComplex) * B_size);
  posix_memalign((void **)&C_h, ALIGN, sizeof(hipFloatComplex) * C_size);

  hipFloatComplex *A_d;
  hipFloatComplex *B_d;
  hipFloatComplex *C1_d;
  hipFloatComplex *C2_d;

  // Device allocation matrix
  hipMalloc((void **)&A_d, sizeof(hipFloatComplex) * A_size);
  hipMalloc((void **)&B_d, sizeof(hipFloatComplex) * B_size);
  hipMalloc((void **)&C1_d, sizeof(hipFloatComplex) * C_size);
  hipMalloc((void **)&C2_d, sizeof(hipFloatComplex) * C_size);

  dim3 gridA((A_size + BLKSIZE - 1) / BLKSIZE);
  dim3 gridB((B_size + BLKSIZE - 1) / BLKSIZE);
  dim3 gridC((C_size + BLKSIZE - 1) / BLKSIZE);
  dim3 block(BLKSIZE);

  hipEventRecord(start, 0);

  fillMatrix<<<gridA, block>>>(A_d, A_size);
  fillMatrix<<<gridB, block>>>(B_d, B_size);
  fillMatrix<<<gridC, block>>>(C1_d, C_size);
  fillMatrix<<<gridC, block>>>(C2_d, C_size);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed, start, stop);
  elapsed /= 1000.0f;

  printf("[DEBUG] Allocation time: %fs\n", elapsed);

  hipComplex alpha = make_hipFloatComplex(1.0f, 1.0f);
  hipComplex beta  = make_hipFloatComplex(0.0f, 0.0f);

  hipEventRecord(start, 0);

  stat = hipblasCgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K,
      &alpha, A_d, K,
      B_d, N, &beta,
      C1_d, N);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed, start, stop);
  elapsed /= 1000.0f;

  printf("hipblasCgemm() taken: %fs\n", elapsed);

  checkError(stat, "\ncublasCgemm() failed!\n");

  //// cublasGemmEX (Tensor Core) ////

  hipEventRecord(start, 0);

  stat = hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K,
      &alpha, A_d, HIP_C_32F, K,
      B_d, HIP_C_32F, N, &beta,
      C2_d, HIP_C_32F, N, HIP_C_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed, start, stop);
  elapsed /= 1000.0f;

  printf("hipblasGemmEx() taken: %fs\n\n", elapsed);

  //// end Tensor Core section ////

  //// Validation section ////
  checkMatrix<<<gridC, block>>>(C1_d, C2_d, C_size, &isValid);
  bool isValidH;
  hipMemcpyFromSymbol(&isValid, HIP_SYMBOL(&isValidH), sizeof(bool));

  if (isValidH) {
    printf("Matrix C1 and C2 are equal\n\n");
  } else {
    printf("Matrix C1 and C2 are not equal\n\n");
  }

  // Deallocation

  hipblasDestroy(handle);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  free(A_h);
  free(B_h);
  free(C_h);

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C1_d);
  hipFree(C2_d);

  printf("End success");

  return EXIT_SUCCESS;
}
