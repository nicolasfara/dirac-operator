
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <mma.h>
#include <hip/hip_complex.h>

#define WARP_SIZE 32
#define BLKSIZE 1024

using namespace nvcuda;

__global__ void dot_wmma16x16(half *a, half *b, float *c)
{
  wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
  wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
  wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;
  wmma::load_matrix_sync(a_frag, a, 16);
  wmma::load_matrix_sync(b_frag, b, 16);
  wmma::fill_fragment(c_frag, 0.0f);
  wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
  wmma::store_matrix_sync(c, c_frag, 16, wmma::mem_row_major);
}

__global__ void mat_sub(half *a, half *b, half *res, const unsigned size)
{
  const unsigned i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < size) {
    res[i] = a[i] - b[i];
  }
}

__global__ void mat_add(half *a, half *b, half *res, const unsigned size)
{
  const unsigned i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < size) {
    res[i] = a[i] + b[i];
  }
}

__global__ void fill_zero(half *re, half *im, float *c)
{
  const unsigned i = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < 16 && j < 16) {
    re[j + 16*i] = __float2half(0.0f);
    im[j + 16*i] = __float2half(0.0f);
    c[j + 16*i] = 0.0f;
  }
}

__host__ __device__ static inline void _sub_matrix_real(half *m, hipDoubleComplex *a)
{
  *m      = __float2half((float) hipCreal(a[0]));
  *(m+1)  = __float2half((float) hipCreal(a[1]));
  *(m+2)  = __float2half((float) hipCreal(a[2]));

  *(m+16) = __float2half((float) hipCreal(a[3]));
  *(m+17) = __float2half((float) hipCreal(a[4]));
  *(m+18) = __float2half((float) hipCreal(a[5]));
  
  *(m+32) = __float2half((float) hipCreal(a[6]));
  *(m+33) = __float2half((float) hipCreal(a[7]));
  *(m+34) = __float2half((float) hipCreal(a[8]));
}

__host__ __device__ static inline void _sub_matrix_imag(half *m, hipDoubleComplex *a)
{
  *m      = __float2half((float) hipCimag(a[0]));
  *(m+1)  = __float2half((float) hipCimag(a[1]));
  *(m+2)  = __float2half((float) hipCimag(a[2]));

  *(m+16) = __float2half((float) hipCimag(a[3]));
  *(m+17) = __float2half((float) hipCimag(a[4]));
  *(m+18) = __float2half((float) hipCimag(a[5]));
  
  *(m+32) = __float2half((float) hipCimag(a[6]));
  *(m+33) = __float2half((float) hipCimag(a[7]));
  *(m+34) = __float2half((float) hipCimag(a[8]));
}

__host__ __device__ static inline void _sub_vec_real(half *v, hipDoubleComplex *a)
{
  *v      = __float2half((float) hipCreal(a[0]));
  *(v+16) = __float2half((float) hipCreal(a[1]));
  *(v+32) = __float2half((float) hipCreal(a[2]));
}

__host__ __device__ static inline void _sub_vec_imag(half *v, hipDoubleComplex *a)
{
  *v      = __float2half((float) hipCimag(a[0]));
  *(v+16) = __float2half((float) hipCimag(a[1]));
  *(v+32) = __float2half((float) hipCimag(a[2]));
}

__global__ void compose_matrix(half *t_mat, half *t_vec, hipDoubleComplex *mat, hipDoubleComplex *vec)
{
  const unsigned gi = threadIdx.x + blockIdx.x * blockDim.x;
  if (gi == 0)   _sub_matrix_real(t_mat, mat);
  if (gi == 51)  _sub_matrix_imag(t_mat + 51, mat);
  if (gi == 102)  _sub_matrix_real(t_mat + 102, mat);
  if (gi == 153) _sub_matrix_imag(t_mat + 153, mat);

  if (gi == 0)   _sub_vec_real(t_vec, vec);
  if (gi == 51)  _sub_vec_imag(t_vec + 51, vec);
  if (gi == 102)  _sub_vec_real(t_vec + 102, vec);
  if (gi == 153) _sub_vec_imag(t_vec + 153, vec);
}

void complex_mma(hipDoubleComplex *mat, hipDoubleComplex *vec, const unsigned size)
{
  const size_t mat_size = 16 * 16;

  half *t_mat;
  half *t_vec;
  float *t_res;

  hipMalloc((void **)& t_mat, sizeof(half) * mat_size);
  hipMalloc((void **)& t_vec, sizeof(half) * mat_size);
  hipMalloc((void **)& t_res, sizeof(float) * mat_size);

  dim3 blockDim(BLKSIZE, BLKSIZE);
  dim3 gridDim((16 + BLKSIZE - 1) / 16, (16 + BLKSIZE - 1) / 16);

  fill_zero<<<gridDim, blockDim>>>(t_mat, t_vec, t_res);
  compose_matrix<<<1, mat_size>>>(t_mat, t_vec, mat, vec);

  half *h_t_mat = (half *) malloc(sizeof(half) * mat_size);
  hipMemcpy(h_t_mat, t_mat, sizeof(half) *mat_size, hipMemcpyDeviceToHost);
  printf("Composed matrix:\n");
  for (unsigned i = 0; i < 16; i++) {
    for (unsigned j = 0; j < 16; j++) {
      printf("%.1f ", __half2float(h_t_mat[j + 16*i]));
    }
    printf("\n");
  }
      

  dot_wmma16x16<<<1, WARP_SIZE>>>(t_mat, t_vec, t_res);

  float *p_res = (float *) malloc(sizeof(float) * mat_size);
  hipMemcpy(p_res, t_res, sizeof(float) * mat_size, hipMemcpyDeviceToHost);

  for (unsigned i = 0; i < 16; i++){
    for (unsigned j = 0; j < 16; j++){
      printf("%.2f ", p_res[j + 16*i]);
    }
    printf("\n");
  }

  // Padding matrix to be used with tensor core
  //matrix_padding_16x16<<<gridDim, blockDim>>>(a, a_re, a_im);
  //vector_padding_16x16<<<gridDim, blockDim>>>(b, b_re, b_im);

  hipFree(t_mat);
  hipFree(t_vec);
  hipFree(t_res);
}

__global__ void fill_matrix(half *m, const unsigned size)
{
  const unsigned i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < size) {
    m[i] = __float2half((float) i);
  }
}

int main(int argc, char **argv)
{

  size_t mat_size = 16 * 16;

  half *a_h;
  half *b_h;
  float *c_h;

  //posix_memalign((void **)&a_h, 128, mat_size * sizeof(half));
  //posix_memalign((void **)&b_h, 128, mat_size * sizeof(half));
  //posix_memalign((void **)&c_h, 128, mat_size * sizeof(half));

  a_h = (half *) malloc(sizeof(half) * mat_size);
  b_h = (half *) malloc(sizeof(half) * mat_size);
  c_h = (float *) malloc(sizeof(float) * mat_size);

  for (unsigned i = 0; i < mat_size; i++) {
    a_h[i] = __float2half((float) i);
    printf("%f ", __half2float(a_h[i]));
  }
  printf("\n\n");
  for (unsigned i = 0; i < mat_size; i++) {
    b_h[i] = __float2half((float)  1);
    printf("%f ", __half2float(b_h[i]));
  }
  printf("\n\n");
  for (unsigned i = 0; i < mat_size; i++) {
    c_h[i] =  0.0f;
    printf("%f ", c_h[i]);
  }
  printf("\n\n");

  half *a_d;
  half *b_d;
  float *c_d;

  hipMalloc((void **)&a_d, sizeof(half) * mat_size);
  hipMalloc((void **)&b_d, sizeof(half) * mat_size);
  hipMalloc((void **)&c_d, sizeof(float) * mat_size);

  hipMemcpy(a_d, a_h, sizeof(half) * mat_size, hipMemcpyHostToDevice);
  hipMemcpy(b_d, b_h, sizeof(half) * mat_size, hipMemcpyHostToDevice);
  hipMemcpy(c_d, c_h, sizeof(float) * mat_size, hipMemcpyHostToDevice);

  dot_wmma16x16<<<1, 32>>>(a_d, b_d, c_d);

  hipMemcpy(c_h, c_d, sizeof(float) * mat_size, hipMemcpyDeviceToHost);

  printf("\n\n");
  for (unsigned i = 0; i < mat_size; i++) {
    printf("%f ", __half2float(c_h[i]));
  }

  printf("\n\nLets cmon\n\n");

  hipDoubleComplex *mat = (hipDoubleComplex *) malloc(sizeof(hipDoubleComplex) * 9);
  hipDoubleComplex *vec = (hipDoubleComplex *) malloc(sizeof(hipDoubleComplex) * 3);

  for (unsigned i = 0; i < 9; i++)
    mat[i] = make_hipDoubleComplex((double) 1, (double) 1);

  for (unsigned i = 0; i < 3; i++)
    vec[i] = make_hipDoubleComplex((double) 1, (double) 1);

  hipDoubleComplex *d_mat, *d_vec;
  hipMalloc((void **)&d_mat, sizeof(hipDoubleComplex) * 9);
  hipMalloc((void **)&d_vec, sizeof(hipDoubleComplex) * 3);
  hipMemcpy(d_mat, mat, sizeof(mat[0]) * 9, hipMemcpyHostToDevice);
  hipMemcpy(d_vec, vec, sizeof(vec[0]) * 3, hipMemcpyHostToDevice);

  printf("compute...\n\n");

  complex_mma(d_mat, d_vec, 3);

  free(c_h);
  free(mat);
  free(vec);

  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);
  hipFree(d_mat);
  hipFree(d_vec);

  return EXIT_SUCCESS;
}
