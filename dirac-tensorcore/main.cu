#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#include "common-cuda.h"

__host__ __device__ static __inline__ void mat_vec_mul( const __restrict su3_soa * const matrix,
                                                        const int idx_mat,
                                                        const int eta,
                                                        const __restrict vec3_soa * const in_vect,
                                                        const int idx_vect,
                                                        __restrict vec3 * const out_vect) {

  hipDoubleComplex vec0 = in_vect->c0[idx_vect];
  hipDoubleComplex vec1 = in_vect->c1[idx_vect];
  hipDoubleComplex vec2 = in_vect->c2[idx_vect];

  hipDoubleComplex mat00 = matrix->r0.c0[idx_mat];
  hipDoubleComplex mat01 = matrix->r0.c1[idx_mat];
  hipDoubleComplex mat02 = matrix->r0.c2[idx_mat];

  hipDoubleComplex mat10 = matrix->r1.c0[idx_mat];
  hipDoubleComplex mat11 = matrix->r1.c1[idx_mat];
  hipDoubleComplex mat12 = matrix->r1.c2[idx_mat];

#ifdef READROW3
// Load 3rd matrix row from global memory
  hipDoubleComplex mat20 = matrix->r2.c0[idx_mat];
  hipDoubleComplex mat21 = matrix->r2.c1[idx_mat];
  hipDoubleComplex mat22 = matrix->r2.c2[idx_mat];
#else
//Compute 3rd matrix row from the first two
  hipDoubleComplex mat20 = hipConj( hipCsub( hipCmul( mat01, mat12 ), hipCmul( mat02, mat11) ) );
  hipDoubleComplex mat21 = hipConj( hipCsub( hipCmul( mat02, mat10 ), hipCmul( mat00, mat12) ) ); 
  hipDoubleComplex mat22 = hipConj( hipCsub( hipCmul( mat00, mat11 ), hipCmul( mat01, mat10) ) );
#endif

//Multiply 3rd row by eta
  mat20 = make_hipDoubleComplex(hipCreal(mat20)*eta, hipCimag(mat20)*eta);
  mat21 = make_hipDoubleComplex(hipCreal(mat21)*eta, hipCimag(mat21)*eta);
  mat22 = make_hipDoubleComplex(hipCreal(mat22)*eta, hipCimag(mat22)*eta);

  out_vect->c0 = hipCadd( hipCadd( hipCmul( mat00, vec0 ),
                                 hipCmul( mat01, vec1 )),
                                 hipCmul( mat02, vec2 ));

  out_vect->c1 = hipCadd( hipCadd( hipCmul( mat10, vec0 ),
                                 hipCmul( mat11, vec1 )),
                                 hipCmul( mat12, vec2 ));

  out_vect->c2 = hipCadd( hipCadd( hipCmul( mat20, vec0 ),
                                 hipCmul( mat21, vec1 )),
                                 hipCmul( mat22, vec2 ));

}

__host__ __device__ static __inline__ void conjmat_vec_mul( const __restrict su3_soa * const matrix,
                                                            const int idx_mat,
                                                            const int eta,
                                                            const __restrict vec3_soa * const in_vect,
                                                            const int idx_vect,
                                                            __restrict vec3 * const out_vect) {

  hipDoubleComplex vec0 = in_vect->c0[idx_vect];
  hipDoubleComplex vec1 = in_vect->c1[idx_vect];
  hipDoubleComplex vec2 = in_vect->c2[idx_vect];

  hipDoubleComplex mat00 = matrix->r0.c0[idx_mat];
  hipDoubleComplex mat01 = matrix->r0.c1[idx_mat];
  hipDoubleComplex mat02 = matrix->r0.c2[idx_mat];

  hipDoubleComplex mat10 = matrix->r1.c0[idx_mat];
  hipDoubleComplex mat11 = matrix->r1.c1[idx_mat];
  hipDoubleComplex mat12 = matrix->r1.c2[idx_mat];

#ifdef READROW3
// Load 3rd matrix row from global memory
//  hipDoubleComplex mat20 = matrix->r2.c0[idx_mat];
//  hipDoubleComplex mat21 = matrix->r2.c1[idx_mat];
//  hipDoubleComplex mat22 = matrix->r2.c2[idx_mat];
#else
//Compute 3rd matrix row from the first two
  hipDoubleComplex mat20 = hipConj( hipCsub( hipCmul( mat01, mat12 ), hipCmul( mat02, mat11) ) );
  hipDoubleComplex mat21 = hipConj( hipCsub( hipCmul( mat02, mat10 ), hipCmul( mat00, mat12) ) );
  hipDoubleComplex mat22 = hipConj( hipCsub( hipCmul( mat00, mat11 ), hipCmul( mat01, mat10) ) );
#endif

//Multiply 3rd row by eta
  mat20 = make_hipDoubleComplex(hipCreal(mat20)*eta, hipCimag(mat20)*eta);
  mat21 = make_hipDoubleComplex(hipCreal(mat21)*eta, hipCimag(mat21)*eta);
  mat22 = make_hipDoubleComplex(hipCreal(mat22)*eta, hipCimag(mat22)*eta);

  out_vect->c0 = hipCadd( hipCadd( hipCmul( hipConj(mat00), vec0 ),
                                 hipCmul( hipConj(mat10), vec1 )),
                                 hipCmul( hipConj(mat20), vec2 ));

  out_vect->c1 = hipCadd( hipCadd( hipCmul( hipConj(mat01), vec0 ),
                                 hipCmul( hipConj(mat11), vec1 )),
                                 hipCmul( hipConj(mat21), vec2 ));

  out_vect->c2 = hipCadd( hipCadd( hipCmul( hipConj(mat02), vec0 ),
                                 hipCmul( hipConj(mat12), vec1 )),
                                 hipCmul( hipConj(mat22), vec2 ));

}

__host__ __device__ static __inline__ vec3 sumResult ( vec3 aux, vec3 aux_tmp) {

  aux.c0 = hipCadd ( aux.c0, aux_tmp.c0);
  aux.c1 = hipCadd ( aux.c1, aux_tmp.c1);
  aux.c2 = hipCadd ( aux.c2, aux_tmp.c2);

  return aux;

}

__host__ __device__ static __inline__ vec3 subResult ( vec3 aux, vec3 aux_tmp) {

  aux.c0 = hipCsub ( aux.c0, aux_tmp.c0);
  aux.c1 = hipCsub ( aux.c1, aux_tmp.c1);
  aux.c2 = hipCsub ( aux.c2, aux_tmp.c2);

  return aux;

}

__global__ void Deo(const __restrict su3_soa * const u, __restrict vec3_soa * const out, const __restrict vec3_soa * const in) {

  int x, y, z, t, xm, ym, zm, tm, xp, yp, zp, tp, idxh, eta; //, idx;

  vec3 aux_tmp;
  vec3 aux;         

  idxh = ((blockIdx.z * blockDim.z + threadIdx.z) * nxh * ny)                                                             
       + ((blockIdx.y * blockDim.y + threadIdx.y) * nxh)                                                                 
       +  (blockIdx.x * blockDim.x + threadIdx.x); // idxh = snum(x,y,z,t)   

//  idx = 2*idxh;
//  t = (idx / vol3) % nt;
//  z = (idx / vol2) % nz;
//  y =   (blockIdx.y * blockDim.y + threadIdx.y);
//  x = 2*(blockIdx.x * blockDim.x + threadIdx.x) + ((y+z+t) % 2);

  t =   (blockIdx.z * blockDim.z + threadIdx.z) / nz;
  z =   (blockIdx.z * blockDim.z + threadIdx.z) % nz;
  y =   (blockIdx.y * blockDim.y + threadIdx.y);
  x = 2*(blockIdx.x * blockDim.x + threadIdx.x) + ((y+z+t) & 0x1);

  xm = x - 1;
  xm = xm + (((xm >> 31) & 0x1) * nx);
  ym = y -1;
  ym = ym + (((ym >> 31) & 0x1) * ny);
  zm = z -1;
  zm = zm + (((zm >> 31) & 0x1) * nz);
  tm = t -1;
  tm = tm + (((tm >> 31) & 0x1) * nt);

  xp = (x+1);
  xp *= (((xp-nx) >> 31) & 0x1);
  yp = (y+1);
  yp *= (((yp-ny) >> 31) & 0x1);
  zp = (z+1);
  zp *= (((zp-nz) >> 31) & 0x1);
  tp = (t+1);
  tp *= (((tp-nt) >> 31) & 0x1);

  eta = 1;
// mat_vec_mul( &(u_work[snum(x,y,z,t)       ]), &(in[snum(xp,y,z,t)]), &aux_tmp );
  mat_vec_mul( &u[0], idxh, eta, in, snum(xp,y,z,t), &aux_tmp );
  aux = aux_tmp;

  eta = 1 - ( 2*(x & 0x1) ); // if (x % 2 = 0) eta = 1 else -1
// mat_vec_mul( &(u_work[snum(x,y,z,t) + size ]), &(in[snum(x,yp,z,t)]), &aux_tmp );
  mat_vec_mul( &u[2], idxh, eta, in, snum(x,yp,z,t), &aux_tmp );
  aux = sumResult(aux, aux_tmp);

  eta = 1 - ( 2*((x+y) & 0x1) );
// mat_vec_mul( &(u_work[snum(x,y,z,t) + size2]), &(in[snum(x,y,zp,t)]), &aux_tmp );
  mat_vec_mul( &u[4], idxh, eta, in, snum(x,y,zp,t), &aux_tmp);
  aux = sumResult(aux, aux_tmp);

  eta = 1 - ( 2*((x+y+z) & 0x1) );
// mat_vec_mul( &(u_work[snum(x,y,z,t) + size3]), &(in[snum(x,y,z,tp)]), &aux_tmp );
  mat_vec_mul( &u[6], idxh, eta, in, snum(x,y,z,tp), &aux_tmp );
  aux = sumResult(aux, aux_tmp);

//////////////////////////////////////////////////////////////////////////////////////////////
    
  eta = 1;
// conjmat_vec_mul( &(u_work[sizeh + snum(xm,y,z,t)      ]), &(in[ snum(xm,y,z,t) ]), &aux_tmp );
  conjmat_vec_mul( &u[1], snum(xm,y,z,t), eta, in, snum(xm,y,z,t), &aux_tmp );
  aux = subResult(aux, aux_tmp);

  eta = 1 - ( 2*(x & 0x1) );
// conjmat_vec_mul( &(u_work[sizeh + snum(x,ym,z,t) + size ]), &(in[ snum(x,ym,z,t) ]), &aux_tmp );
  conjmat_vec_mul( &u[3], snum(x,ym,z,t), eta, in, snum(x,ym,z,t), &aux_tmp );
  aux = subResult(aux, aux_tmp);

  eta = 1 - ( 2*((x+y) & 0x1) );
// conjmat_vec_mul( &(u_work[sizeh + snum(x,y,zm,t) + size2]), &(in[ snum(x,y,zm,t) ]), &aux_tmp );
  conjmat_vec_mul( &u[5], snum(x,y,zm,t), eta, in, snum(x,y,zm,t), &aux_tmp );
  aux = subResult(aux, aux_tmp);

  eta = 1 - ( 2*((x+y+z) & 0x1) );
// conjmat_vec_mul( &(u_work[sizeh + snum(x,y,z,tm) + size3]), &(in[ snum(x,y,z,tm) ]), &aux_tmp );
  conjmat_vec_mul( &u[7], snum(x,y,z,tm), eta, in, snum(x,y,z,tm), &aux_tmp );
  aux = subResult(aux, aux_tmp);

//////////////////////////////////////////////////////////////////////////////////////////////

  out->c0[idxh] = make_hipDoubleComplex(hipCreal(aux.c0)*0.5, hipCimag(aux.c0)*0.5);
  out->c1[idxh] = make_hipDoubleComplex(hipCreal(aux.c1)*0.5, hipCimag(aux.c1)*0.5);
  out->c2[idxh] = make_hipDoubleComplex(hipCreal(aux.c2)*0.5, hipCimag(aux.c2)*0.5);

}

__global__ void Doe(const __restrict su3_soa * const u, __restrict vec3_soa * const out, const __restrict vec3_soa * const in) {

  int x, y, z, t, xm, ym, zm, tm, xp, yp, zp, tp, idxh, eta; //, idx;

  vec3 aux_tmp;
  vec3 aux;

  idxh = ((blockIdx.z * blockDim.z + threadIdx.z) * nxh * ny)                                                             
       + ((blockIdx.y * blockDim.y + threadIdx.y) * nxh)                                                                 
       +  (blockIdx.x * blockDim.x + threadIdx.x); // idxh = snum(x,y,z,t)   

//  idx = 2*idxh;
//  t = (idx / vol3) % nt;
//  z = (idx / vol2) % nz;
//  y =   (blockIdx.y * blockDim.y + threadIdx.y);
//  x = 2*(blockIdx.x * blockDim.x + threadIdx.x) + ((y+z+t+1) % 2);

  t =   (blockIdx.z * blockDim.z + threadIdx.z) / nz;
  z =   (blockIdx.z * blockDim.z + threadIdx.z) % nz;
  y =   (blockIdx.y * blockDim.y + threadIdx.y);
  x = 2*(blockIdx.x * blockDim.x + threadIdx.x) + ((y+z+t+1) & 0x1);

  xm = x - 1;
  xm = xm + (((xm >> 31) & 0x1) * nx);
  ym = y -1;
  ym = ym + (((ym >> 31) & 0x1) * ny);
  zm = z -1;
  zm = zm + (((zm >> 31) & 0x1) * nz);
  tm = t -1;
  tm = tm + (((tm >> 31) & 0x1) * nt);

  xp = (x+1);
  xp *= (((xp-nx) >> 31) & 0x1);
  yp = (y+1);
  yp *= (((yp-ny) >> 31) & 0x1);
  zp = (z+1);
  zp *= (((zp-nz) >> 31) & 0x1);
  tp = (t+1);
  tp *= (((tp-nt) >> 31) & 0x1);

  eta = 1;
// mat_vec_mul( &(u_work[snum(x,y,z,t) + sizeh      ]), &(in[ snum(xp,y,z,t) ]), &aux_tmp );
  mat_vec_mul( &u[1], idxh, eta, in, snum(xp,y,z,t), &aux_tmp );
  aux = aux_tmp;

  eta = 1 - ( 2*(x & 0x1) );
// mat_vec_mul( &(u_work[snum(x,y,z,t) + sizeh + size ]), &(in[ snum(x,yp,z,t) ]), &aux_tmp );
  mat_vec_mul( &u[3], idxh, eta, in, snum(x,yp,z,t), &aux_tmp );
  aux = sumResult(aux, aux_tmp);

  eta = 1 - ( 2*((x+y) & 0x1) );
// mat_vec_mul( &( u_work[snum(x,y,z,t) + sizeh + size2]), &(in[ snum(x,y,zp,t) ]), &aux_tmp );
  mat_vec_mul( &u[5], idxh, eta, in, snum(x,y,zp,t), &aux_tmp );
  aux = sumResult(aux, aux_tmp);

  eta = 1 - ( 2*((x+y+z) & 0x1) );
// mat_vec_mul( &(u_work[snum(x,y,z,t) + sizeh + size3]), &(in[ snum(x,y,z,tp) ]), &aux_tmp );
  mat_vec_mul( &u[7], idxh, eta, in, snum(x,y,z,tp), &aux_tmp );
  aux = sumResult(aux, aux_tmp);

//////////////////////////////////////////////////////////////////////////////////////////////

  eta = 1;
// conjmat_vec_mul( &(u_work[snum(xm,y,z,t)      ]), &(in[ snum(xm,y,z,t) ]), &aux_tmp );
  conjmat_vec_mul( &u[0], snum(xm,y,z,t), eta, in, snum(xm,y,z,t), &aux_tmp );
  aux = subResult(aux, aux_tmp);

  eta = 1 - ( 2*(x & 0x1) );
// conjmat_vec_mul( &(u_work[snum(x,ym,z,t) + size ]), &(in[ snum(x,ym,z,t) ]), &aux_tmp );
  conjmat_vec_mul( &u[2], snum(x,ym,z,t), eta, in, snum(x,ym,z,t), &aux_tmp );
  aux = subResult(aux, aux_tmp);

  eta = 1 - ( 2*((x+y) & 0x1) );
// conjmat_vec_mul( &(u_work[snum(x,y,zm,t) + size2]), &(in[ snum(x,y,zm,t) ]), &aux_tmp );
  conjmat_vec_mul( &u[4], snum(x,y,zm,t), eta, in, snum(x,y,zm,t), &aux_tmp );
  aux = subResult(aux, aux_tmp);

  eta = 1 - ( 2*((x+y+z) & 0x1) );
// conjmat_vec_mul( &(u_work[snum(x,y,z,tm) + size3]), &(in[ snum(x,y,z,tm) ]), &aux_tmp );
  conjmat_vec_mul( &u[6], snum(x,y,z,tm), eta, in, snum(x,y,z,tm), &aux_tmp );
  aux = subResult(aux, aux_tmp);

//////////////////////////////////////////////////////////////////////////////////////////////

  out->c0[idxh] = make_hipDoubleComplex(hipCreal(aux.c0)*0.5, hipCimag(aux.c0)*0.5);
  out->c1[idxh] = make_hipDoubleComplex(hipCreal(aux.c1)*0.5, hipCimag(aux.c1)*0.5);
  out->c2[idxh] = make_hipDoubleComplex(hipCreal(aux.c2)*0.5, hipCimag(aux.c2)*0.5);

}


int main() {

  int i;
  struct timeval t0, t1;
  double dt_tot = 0.0;

  dim3 dimBlockK1 (DIM_BLOCK_X, DIM_BLOCK_Y, DIM_BLOCK_Z);
//  dim3 dimGridK1  ((nx*ny*nz*nt/2)/DIM_BLOCK_X, 1, 1 );
  dim3 dimGridK1  ((nx/2)/DIM_BLOCK_X, ny/DIM_BLOCK_Y, (nz*nt)/DIM_BLOCK_Z );

  if ( ((nx % 2) != 0) || (((nx/2) % DIM_BLOCK_X) != 0) ) {
    fprintf(stderr, "ERROR: nx should be even and nx/2 should be divisible by DIM_BLOCK_X.");
    return -1;
  }

  su3_soa * u_h;
  half * u_ht[8];
  vec3_soa * fermion1_h;
  half * fermion1_ht;
  vec3_soa * fermion2_h;
  half * fermion2_ht;

  // 8 = number of directions times 2 (even/odd)
  // no_links = sizeh * 8
  posix_memalign((void **)&u_h,        ALIGN, 8*sizeof(su3_soa));
  for (unsigned i=0; i<8; i++)
    posix_memalign((void **)&u_ht[i],       ALIGN, (sizeh/8)*256*sizeof(half));
  posix_memalign((void **)&fermion1_h, ALIGN, sizeof(vec3_soa));
  posix_memalign((void **)&fermion1_ht,ALIGN, (sizeh/8)*256*sizeof(half) );
  posix_memalign((void **)&fermion2_h, ALIGN, sizeof(vec3_soa));
  posix_memalign((void **)&fermion2_ht,ALIGN, (sizeh/8)*256*sizeof(half));

//  printf("Sizeof su3_soa   is: %d \n", sizeof(su3_soa));
//  printf("Sizeof su3_soa_d is: %d \n", sizeof(su3_soa_d));

  su3_soa * u_d;
  half * u_dt[8];
  vec3_soa * fermion1_d;
  half * fermion1_dt;
  vec3_soa * fermion2_d;
  half * fermion2_dt;

  hipMalloc ((void**)&u_d, 8*sizeof(su3_soa));
  checkCUDAError("Allocating u_d");
  for (unsigned i=0; i < 8; i++)
    hipMalloc ((void**)&u_dt[i], (sizeh/8)*256*sizeof(half));
  checkCUDAError("Allocating u_dt");
  hipMalloc ((void**)&fermion1_d, sizeof(vec3_soa));
  checkCUDAError("Allocating fermion1_d");
  hipMalloc ((void**)&fermion1_dt, (sizeh/8)*256*sizeof(half));
  checkCUDAError("Allocating fermion1_dt");
  hipMalloc ((void**)&fermion2_d, sizeof(vec3_soa));
  checkCUDAError("Allocating fermion2_d");
  hipMalloc ((void**)&fermion2_dt, (sizeh/8)*256*sizeof(half));
  checkCUDAError("Allocating fermion2_dt");


if ((nx == 32) && (ny == 32) && (nz == 32) && (nt == 32)) {
  loadSu3FromFileNew( u_h, "gaugeconf_save_32_4");
  loadFermionFromFileNew(fermion1_h, "test_fermion_32_4");
} else if ((nx == 16) && (ny == 16) && (nz == 16) && (nt == 16)) {
  loadSu3FromFile( u_h, "TestConf_16_4.cnf");
  for (unsigned i=0; i<8; i++)
    Su3Mapper(u_h[i], u_ht[i]);
  printf("Mapper Su3_soa\n");
  loadFermionFromFile(fermion1_h, "StartFermion_16_4.fer");
  fermionMapper(fermion1_h, fermion1_ht);
  printf("fermion mapper\n");
} else {
  fprintf(stdout, "Lattice not available... \n");
  exit(1);
}

  printMappedSu3_soa(u_ht[0], u_h[0]);
  printMappedVec3_soa(fermion1_ht, *fermion1_h);

  hipMemcpy( u_d, u_h, 8*sizeof(su3_soa), hipMemcpyHostToDevice );
  checkCUDAError("Copying u_d to device");
  hipMemcpy( fermion1_d, fermion1_h, sizeof(vec3_soa), hipMemcpyHostToDevice );
  checkCUDAError("Copying fermion1_d to device");
  hipMemcpy( fermion2_d, fermion2_h, sizeof(vec3_soa), hipMemcpyHostToDevice );
  checkCUDAError("Copying fermion2_d to device");

  // Prefer larger L1 cache than shared mem
   hipDeviceSetCacheConfig(hipFuncCachePreferL1);
  // Prefer larger shared mem than L1 cache
  // hipDeviceSetCacheConfig(hipFuncCachePreferShared);

  gettimeofday ( &t0, NULL );

  for (i = 0; i < NITER; i++) {
    Deo<<< dimGridK1, dimBlockK1 >>>( u_d, fermion2_d, fermion1_d);
    checkCUDAError("Running kernel Deo");
    //hipDeviceSynchronize();
    //checkCUDAError("Cuda synch after Deo");
    Doe<<< dimGridK1, dimBlockK1 >>>( u_d, fermion1_d, fermion2_d);
    checkCUDAError("Running kernel Doe");
    //hipDeviceSynchronize();
    //checkCUDAError("Cuda synch after Doe");
  }

  hipDeviceSynchronize();
  gettimeofday ( &t1, NULL );

//  hipMemcpy( fermion1_h, fermion2_d, sizeof(vec3_soa), hipMemcpyDeviceToHost );
  hipMemcpy( fermion1_h, fermion1_d, sizeof(vec3_soa), hipMemcpyDeviceToHost );
  checkCUDAError("Copying fermion1_d to host");

  dt_tot = (double)(t1.tv_sec - t0.tv_sec) + ((double)(t1.tv_usec - t0.tv_usec)/1.0e6);

  printf("TOTAL Exec time:          Tot time: % 3.2f sec    Avg: % 3.02f ms   Avg/site: % 3.02f ns\n",
          dt_tot, \
          (dt_tot/NITER)*(1.0e3),
          ((dt_tot/NITER)/size)*(1.0e9) );

  writeFermionToFile(fermion1_h, "EndFermion.fer");

  free(u_h);
  for (unsigned i=0; i<8; i++)
    free(u_ht[i]);
  free(fermion1_h);
  free(fermion1_ht);
  free(fermion2_h);
  free(fermion2_ht);

  return 0;

}
