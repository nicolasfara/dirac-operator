#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cublas-utility.h"
#include "cuda_utility.h"

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

using namespace std;

int main(int argc, char **argv)
{
  /* Parse input args */
  int mat_side = 3;
  int batch_complex = 10;
  int batch = batch_complex*4;

  if (argc == 3) {
    mat_side = atoi(argv[1]); // Matrix side
    batch_complex = atoi(argv[2]);    // Number of MMA
    batch = batch_complex*4;
  } else {
    fprintf(stderr, "./%s SIDE BATCH", argv[0]);
    return EXIT_FAILURE;
  }

  printf("Processing input args: %d side, %d batch\n\n", mat_side, batch_complex);

  hipblasHandle_t handle;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  checkCublas(hipblasCreate(&handle));
  checkCublas(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));

  half **devPtrA = (half **)malloc(batch_complex * sizeof(*devPtrA));
  half **devPtrB = (half **)malloc(batch_complex * sizeof(*devPtrB));
  half **devPtrC = (half **)malloc(batch_complex * sizeof(*devPtrC));
  half **devPtrA_dev, **devPtrB_dev, **devPtrC_dev;

  for (int i = 0; i < batch ; i++) {
    allocate_matrix((void **)&devPtrA[i], mat_side * mat_side * sizeof(devPtrA[0][0]));
    allocate_matrix((void **)&devPtrB[i], mat_side * mat_side * sizeof(devPtrB[0][0]));
    allocate_matrix((void **)&devPtrC[i], mat_side * mat_side * sizeof(devPtrC[0][0]));
  }

  checkCudaErrors(hipMalloc((void **)&devPtrA_dev, batch_complex * sizeof(*devPtrA)));
  checkCudaErrors(hipMalloc((void **)&devPtrB_dev, batch_complex * sizeof(*devPtrB)));
  checkCudaErrors(hipMalloc((void **)&devPtrC_dev, batch_complex * sizeof(*devPtrC)));
  checkCudaErrors(hipMemcpy(devPtrA_dev, devPtrA, batch_complex * sizeof(*devPtrA), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(devPtrB_dev, devPtrB, batch_complex * sizeof(*devPtrB), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(devPtrC_dev, devPtrC, batch_complex * sizeof(*devPtrC), hipMemcpyHostToDevice));

  fill_matrix(devPtrA[0], mat_side, mat_side);
  fill_matrix(devPtrB[0], mat_side, mat_side);
  fill_matrix(devPtrC[0], mat_side, mat_side);

  /****************** Test without tcu ****************************************/

  checkCudaErrors(hipEventRecord(start, 0));

  mma_batched(handle, mat_side, mat_side, mat_side, devPtrA_dev, devPtrB_dev, devPtrC_dev, batch_complex);
  display_matrix(devPtrC[0], mat_side, mat_side);

  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));
  float elapsed;
  checkCudaErrors(hipEventElapsedTime(&elapsed, start, stop));
  elapsed /= 1000.0f;
  printf("Elapsed WITHOUT TCU:\t %fs\n", elapsed);

  /*************************** Test with TCU **********************************/

  checkCudaErrors(hipEventRecord(start, 0));

  mma_batched_tcu(handle, mat_side, mat_side, mat_side, (void **)devPtrA_dev, (void **)devPtrB_dev, (void **)devPtrC_dev, batch_complex);
  display_matrix(devPtrC[0], mat_side, mat_side);

  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&elapsed, start, stop));
  elapsed /= 1000.0f;
  printf("Elapsed WITH TCU:\t %fs\n", elapsed);

  /************************* COMPLEX SECTION **********************************/

  hipDoubleComplex *mat = (hipDoubleComplex *) malloc(sizeof(hipDoubleComplex) * 9);
  hipDoubleComplex *vec = (hipDoubleComplex *) malloc(sizeof(hipDoubleComplex) * 3);
  hipDoubleComplex *res = (hipDoubleComplex *) malloc(sizeof(hipDoubleComplex) * 3);

  for (unsigned i = 0; i < 9; i++)
    mat[i] = make_hipDoubleComplex((double) 1, (double) 1);

  for (unsigned i = 0; i < 3; i++)
    vec[i] = make_hipDoubleComplex((double) 1, (double) 1);

  hipDoubleComplex *d_mat, *d_vec, *d_res;
  hipMalloc((void **)&d_mat, sizeof(hipDoubleComplex) * 9);
  hipMalloc((void **)&d_vec, sizeof(hipDoubleComplex) * 3);
  hipMalloc((void **)&d_res, sizeof(hipDoubleComplex) * 3);
  hipMemcpy(d_mat, mat, sizeof(mat[0]) * 9, hipMemcpyHostToDevice);
  hipMemcpy(d_vec, vec, sizeof(vec[0]) * 3, hipMemcpyHostToDevice);

  checkCudaErrors(hipEventRecord(start, 0));

  test_3x3matvec(d_mat, d_vec, d_res, batch);

  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&elapsed, start, stop));
  elapsed /= 1000.0f;
  printf("Elapsed complex:\t %fs\n", elapsed);

  return EXIT_SUCCESS;
}
