
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cublas-utility.h"
#include "cuda_utility.h"

using namespace std;

int main(int argc, char **argv)
{
  /* Parse input args */
  int mat_side = 3;
  int batch = 10;

  if (argc == 3) {
    mat_side = atoi(argv[1]); // Matrix side
    batch = atoi(argv[2]);    // Number of MMA
  } else {
    fprintf(stderr, "./%s SIDE BATCH", argv[0]);
    return EXIT_FAILURE;
  }

  printf("Processing input args: %d side, %d batch\n\n", mat_side, batch);

  hipblasHandle_t handle;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);


  checkCublas(hipblasCreate(&handle));
  checkCublas(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));

  hipStream_t *streamArray = (hipStream_t *)malloc(batch * sizeof(hipStream_t *));
  for (int i = 0; i < batch ; i++)
      checkCudaErrors(hipStreamCreate(&streamArray[i]));

  half **devPtrA = (half **)malloc(batch * sizeof(*devPtrA));
  half **devPtrB = (half **)malloc(batch * sizeof(*devPtrB));
  half **devPtrC = (half **)malloc(batch * sizeof(*devPtrC));
  half **devPtrA_dev, **devPtrB_dev, **devPtrC_dev;

  for (int i = 0; i < batch ; i++)
  {
    allocate_matrix((void **)&devPtrA[i], mat_side * mat_side * sizeof(devPtrA[0][0]));
    allocate_matrix((void **)&devPtrB[i], mat_side * mat_side * sizeof(devPtrB[0][0]));
    allocate_matrix((void **)&devPtrC[i], mat_side * mat_side * sizeof(devPtrC[0][0]));
  }

  hipMalloc((void **)&devPtrA_dev, batch * sizeof(*devPtrA));
  hipMalloc((void **)&devPtrB_dev, batch * sizeof(*devPtrB));
  hipMalloc((void **)&devPtrC_dev, batch * sizeof(*devPtrC));
  hipMemcpy(devPtrA_dev, devPtrA, batch * sizeof(*devPtrA), hipMemcpyHostToDevice);
  hipMemcpy(devPtrB_dev, devPtrB, batch * sizeof(*devPtrB), hipMemcpyHostToDevice);
  hipMemcpy(devPtrC_dev, devPtrC, batch * sizeof(*devPtrC), hipMemcpyHostToDevice);


  checkCudaErrors(hipEventRecord(start, 0));

  mma_batched(handle, streamArray, mat_side, mat_side, mat_side, devPtrA_dev, devPtrB_dev, devPtrC_dev, batch);

  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));
  float elapsed;
  checkCudaErrors(hipEventElapsedTime(&elapsed, start, stop));
  elapsed /= 1000.0f;
  printf("Elapsed WITHOUT TCU:\t %fs\n", elapsed);


  checkCudaErrors(hipEventRecord(start, 0));

  mma_batched_tcu(handle, streamArray, mat_side, mat_side, mat_side, (void **)devPtrA_dev, (void **)devPtrB_dev, (void **)devPtrC_dev, batch);

  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&elapsed, start, stop));
  elapsed /= 1000.0f;
  printf("Elapsed WITH TCU:\t %fs\n", elapsed);

  /*************** COMPLEX SECTION **************************/

  hipDoubleComplex *mat = (hipDoubleComplex *) malloc(sizeof(hipDoubleComplex) * 9);
  hipDoubleComplex *vec = (hipDoubleComplex *) malloc(sizeof(hipDoubleComplex) * 3);
  hipDoubleComplex *res = (hipDoubleComplex *) malloc(sizeof(hipDoubleComplex) * 3);

  for (unsigned i = 0; i < 9; i++)
    mat[i] = make_hipDoubleComplex((double) 1, (double) 1);

  for (unsigned i = 0; i < 3; i++)
    vec[i] = make_hipDoubleComplex((double) 1, (double) 1);

  hipDoubleComplex *d_mat, *d_vec, *d_res;
  hipMalloc((void **)&d_mat, sizeof(hipDoubleComplex) * 9);
  hipMalloc((void **)&d_vec, sizeof(hipDoubleComplex) * 3);
  hipMalloc((void **)&d_res, sizeof(hipDoubleComplex) * 3);
  hipMemcpy(d_mat, mat, sizeof(mat[0]) * 9, hipMemcpyHostToDevice);
  hipMemcpy(d_vec, vec, sizeof(vec[0]) * 3, hipMemcpyHostToDevice);

  checkCudaErrors(hipEventRecord(start, 0));

  test_3x3matvec(d_mat, d_vec, d_res, batch);

  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&elapsed, start, stop));
  elapsed /= 1000.0f;
  printf("Elapsed WITH TCU:\t %fs\n", elapsed);

  return EXIT_SUCCESS;
}
